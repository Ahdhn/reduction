#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits>
#include <numeric>
#include <random>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>

#include "helper.h"

__device__ __inline__ void reset_res(volatile int* d_flag,
                                     int*          d_res,
                                     int           init_value)
{
    __threadfence();
    while (true) {
        __threadfence();

        int prv = ::atomicCAS((int*)d_flag, 0, 1);

        // means some other thread set the value
        if (prv == 2) {
            break;
        }

        // means this is the first threads to set the flag
        if (prv == 0) {
            __threadfence();
            d_res[0] = init_value;            
            __threadfence();
            // set the flag to 2, so other threads stop spinning
            ::atomicExch((int*)d_flag, 2);
            break;
        }
    }
}

__device__ __inline__ void reset_flag(volatile int* d_counter,
                                      volatile int* d_flag,
                                      int           size)
{
    __threadfence();
    int id = atomicAdd((int*)d_counter, 1);
    if (id == size - 1) {
        // this is the last thread to contrinute so it can reset the counter and
        // the flag
        d_flag[0]    = 0;
        d_counter[0] = 0;
    }
}

__global__ void sum(int*          d_data,
                    int*          d_res,
                    volatile int* d_flag,
                    volatile int* d_counter,
                    int           init_value,
                    int           size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    reset_res(d_flag, d_res, init_value);

    if (idx < size) {
        atomicAdd(d_res, d_data[idx]);
        reset_flag(d_counter, d_flag, size);
    }
}

__global__ void mmin(int*          d_data,
                     int*          d_res,
                     volatile int* d_flag,
                     volatile int* d_counter,
                     int           init_value,
                     int           size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    reset_res(d_flag, d_res, init_value);

    if (idx < size) {
        atomicMin(d_res, d_data[idx]);
        reset_flag(d_counter, d_flag, size);
    }
}


__global__ void mmax(int*          d_data,
                     int*          d_res,
                     volatile int* d_flag,
                     volatile int* d_counter,
                     int           init_value,
                     int           size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    reset_res(d_flag, d_res, init_value);

    if (idx < size) {
        atomicMax(d_res, d_data[idx]);
        reset_flag(d_counter, d_flag, size);
    }
}

void verify_sum(thrust::host_vector<int>&   h_vec,
                thrust::device_vector<int>& d_res)
{

    thrust::host_vector<int> h_res = d_res;

    int sum_val = std::accumulate(h_vec.begin(), h_vec.end(), 0);

    printf("\nres= %d, sum_val= %d", h_res[0], sum_val);
    if (sum_val != h_res[0]) {
        printf("verify_sum FAILED!!!!!!");
    }
}

void verify_min(thrust::host_vector<int>&   h_vec,
                thrust::device_vector<int>& d_res)
{

    thrust::host_vector<int> h_res = d_res;

    int min_val = *std::min_element(h_res.begin(), h_res.end());

    printf("\nres= %d, min_val= %d", h_res[0], min_val);
    if (min_val != h_res[0]) {
        printf("verify_min FAILED!!!!!!");
    }
}

void verify_max(thrust::host_vector<int>&   h_vec,
                thrust::device_vector<int>& d_res)
{

    thrust::host_vector<int> h_res = d_res;

    int max_val = *std::max_element(h_res.begin(), h_res.end());


    printf("\nres= %d, max_val= %d", h_res[0], max_val);
    if (max_val != h_res[0]) {
        printf("verify_max FAILED!!!!!!");
    }
}

int main(int argc, char** argv)
{

    int N = 10000;
    if (argc == 2) {
        N = std::atoi(argv[1]);
    }


    // Generate a thrust::host_vector with random integers
    thrust::host_vector<int> h_vec(N);

    std::iota(h_vec.begin(), h_vec.end(), N / 4);
    std::random_device rd;
    std::mt19937       g(rd());
    std::shuffle(h_vec.begin(), h_vec.end(), g);

    // Move data to the GPU
    thrust::device_vector<int> d_vec = h_vec;

    // result, flag -- uninitialized
    thrust::device_vector<int> d_res(1);
    thrust::device_vector<int> d_flag(1);

    // counter initilized to zero
    thrust::device_vector<int> d_counter(1, 0);


    // Sum kernel launch
    const int threads = 512;
    const int blocks  = DIVIDE_UP(N, threads);

    mmax<<<blocks, threads>>>(thrust::raw_pointer_cast(d_vec.data()),
                              thrust::raw_pointer_cast(d_res.data()),
                              thrust::raw_pointer_cast(d_flag.data()),
                              thrust::raw_pointer_cast(d_counter.data()),
                              std::numeric_limits<int>::lowest(),
                              N);
    verify_max(h_vec, d_res);

    sum<<<blocks, threads>>>(thrust::raw_pointer_cast(d_vec.data()),
                             thrust::raw_pointer_cast(d_res.data()),
                             thrust::raw_pointer_cast(d_flag.data()),
                             thrust::raw_pointer_cast(d_counter.data()),
                             0,
                             N);
    verify_sum(h_vec, d_res);


    mmin<<<blocks, threads>>>(thrust::raw_pointer_cast(d_vec.data()),
                              thrust::raw_pointer_cast(d_res.data()),
                              thrust::raw_pointer_cast(d_flag.data()),
                              thrust::raw_pointer_cast(d_counter.data()),
                              std::numeric_limits<int>::max(),
                              N);
    verify_min(h_vec, d_res);


    CUDA_ERROR(hipDeviceSynchronize());


    return 0;
}
